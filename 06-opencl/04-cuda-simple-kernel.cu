#include <hip/hip_runtime.h>

#include <iostream>

__global__ void simple_kernel(int* buffer) {
    *buffer = 8601;
}

int main() {
    size_t buf_size = sizeof(int);

    int* buffer;
    hipMalloc(&buffer, buf_size);

    simple_kernel<<<1, 1>>>(buffer);

    hipDeviceSynchronize();

    int host_int;
    hipMemcpy(&host_int, buffer, buf_size, hipMemcpyDeviceToHost);

    std::cout << "Result is : " << host_int << '\n';
}
